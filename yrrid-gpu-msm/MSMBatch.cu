/***

Copyright (c) 2023-2024, Yrrid Software, Inc. and Snarkify Inc.  All rights reserved.
Dual licensed under the MIT License or the Apache License, Version 2.0.
See LICENSE for details.

Author(s):  Niall Emmart

***/

#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include "HostFields.cpp"
#include "HostCurves.cpp"
#include "asm.cu"
#include "Chain.cu"
#include "MP.cu"
#include "Types.h"
#include "Support.cu"
#include "MatterLabs.cuh"
#include "BLS12377.cuh"
#include "BLS12377.cu"
#include "BLS12381.cuh"
#include "BLS12381.cu"
#include "PlanningParameters.cu"
#include "PlanningKernels.cu"
#include "PrecomputeKernels.cu"
#include "AccumulateKernels.cu"
#include "ReduceKernels.cu"
#include "MSMRunner.cuh"
#include "MSMRunner.cu"

#define CUDA_CHECK(call) { int localEC=call; if(localEC!=hipSuccess) { printf("\nCall \"" #call "\" failed from %s, line %d, error=%d\n", __FILE__, __LINE__, localEC); exit(1); } }

/********************************************************************************************
 * Possible run options for BLS12377G1:
 *
 *     typedef MSMRunner<BLS12377G1, ACCUMULATION_TWISTED_EDWARDS_XY, 23, 6, 2> Run377;
 *     typedef MSMRunner<BLS12377G1, ACCUMULATION_TWISTED_EDWARDS_XYT, 23, 6, 2> Run377;
 *     typedef MSMRunner<BLS12377G1, ACCUMULATION_EXTENDED_JACOBIAN, 23, 6, 2> Run377;
 *     typedef MSMRunner<BLS12377G1, ACCUMULATION_EXTENDED_JACOBIAN_ML, 23, 6, 2> Run377;
 *
 * Possible run options for BLS12381G1:
 *     typedef MSMRunner<BLS12381G1, ACCUMULATION_EXTENDED_JACOBIAN, 22, 5, 2> Run381;
 *     typedef MSMRunner<BLS12381G1, ACCUMULATION_EXTENDED_JACOBIAN_ML, 22, 5, 2> Run381;
 * 
 ********************************************************************************************/

typedef MSMRunner<BLS12377G1, ACCUMULATION_TWISTED_EDWARDS_XY, 23, 6, 2> Best377;
typedef MSMRunner<BLS12381G1, ACCUMULATION_EXTENDED_JACOBIAN_ML, 22, 6, 2> Best381;

class MSMBatch {
  public:
  uint32_t     curve;
  uint32_t     maxBatchCount;
  uint32_t     maxPointCount;

  Best377*     runner377;
  Best381*     runner381;

  void*        pointMemory;
  void*        scaledPointMemory;
  void*        scalarMemory;
  void*        planningMemory;
  void*        bucketMemory;
  void*        reduceMemory;

  hipEvent_t  eighthReadyEvent;
  hipEvent_t  halfReadyEvent;
  hipEvent_t* scalarReadyEvents;
  hipEvent_t* resultReadyEvents;

  // ok -- this isn't ideal to dispatch to one curve or the other... oh well

  MSMBatch(uint32_t inCurve, uint32_t inMaxBatchCount, uint32_t inMaxPointCount) {
    curve=inCurve;
    maxBatchCount=inMaxBatchCount;
    maxPointCount=inMaxPointCount;

    runner377=new Best377(inMaxPointCount);
    runner381=new Best381(inMaxPointCount);

    pointMemory=NULL;
    scaledPointMemory=NULL;
    scalarMemory=NULL;
    planningMemory=NULL;
    bucketMemory=NULL;
    reduceMemory=NULL;

    scalarReadyEvents=NULL;
    resultReadyEvents=NULL;
  }

  ~MSMBatch() {
    delete runner377;
    delete runner381;
  }

  size_t pointBytesRequired() {
    return curve==377 ? runner377->pointBytesRequired() : runner381->pointBytesRequired();
  }

  size_t planningBytesRequired() {
    return curve==377 ? runner377->planningBytesRequired() : runner381->planningBytesRequired();
  }

  size_t bucketBytesRequired() {
    return curve==377 ? runner377->bucketBytesRequired() : runner381->bucketBytesRequired();
  }

  size_t reduceBytesRequired() {
    return curve==377 ? runner377->reduceBytesRequired() : runner381->reduceBytesRequired();
  }

  int32_t runPointGeneration(hipStream_t stream, void* pointMemory, void* secretScalars, uint32_t pointCount) {
    if(curve==377) 
      return runner377->runPointGeneration(stream, pointMemory, secretScalars, pointCount);
    else if(curve==381)
      return runner381->runPointGeneration(stream, pointMemory, secretScalars, pointCount);
    return -1;
  }

  int32_t runPointPrecompute(hipStream_t stream, void* scaledPointMemory, void* pointMemory, uint32_t pointCount) {
    if(curve==377) 
      return runner377->runPointPrecompute(stream, scaledPointMemory, pointMemory, pointCount);
    else if(curve==381)
      return runner381->runPointPrecompute(stream, scaledPointMemory, pointMemory, pointCount);
    return -1;
  }

  int32_t runPlanning(hipStream_t stream, void* planningMemory, void* scalars, uint32_t startPoint, uint32_t stopPoint) {
    if(curve==377)
      return runner377->runPlanning(stream, planningMemory, scalars, startPoint, stopPoint);
    else if(curve==381)
      return runner381->runPlanning(stream, planningMemory, scalars, startPoint, stopPoint);
    return -1;
  }

  int32_t runPlanning(hipStream_t stream, void* planningMemory, void* scalars, uint32_t pointCount) {
    if(curve==377)
      return runner377->runPlanning(stream, planningMemory, scalars, pointCount);
    else if(curve==381)
      return runner381->runPlanning(stream, planningMemory, scalars, pointCount);
    return -1;
  }

  int32_t runAccumulate(hipStream_t stream, void* bucketMemory, void* planningMemory, void* scaledPointMemory, bool preloaded=false) {
    if(curve==377)
      return runner377->runAccumulate(stream, bucketMemory, planningMemory, scaledPointMemory, preloaded);
    else if(curve==381)
      return runner381->runAccumulate(stream, bucketMemory, planningMemory, scaledPointMemory, preloaded);
    return -1;
  }

  int32_t runReduce(hipStream_t stream, uint32_t* resultPointCount, void* reduceMemory, void* bucketMemory) {
    if(curve==377)
      return runner377->runReduce(stream, resultPointCount, reduceMemory, bucketMemory);
    else if(curve==381)
      return runner381->runReduce(stream, resultPointCount, reduceMemory, bucketMemory);
    return -1;
  }

  int32_t runFinalReduce(void* finalResult, void* cpuReduceMemory, uint32_t resultPointCount) {
    if(curve==377) {
      runner377->runFinalReduce(finalResult, cpuReduceMemory, resultPointCount);
      return 0;
    }
    else if(curve==381) {
      runner381->runFinalReduce(finalResult, cpuReduceMemory, resultPointCount);
      return 0;
    }
    return -1;
  }

};

void* scalarOffset(void* ptr, int32_t scalarCount) {
  int64_t  offset=scalarCount;
  uint8_t* ptr8=(uint8_t*)ptr;

  offset*=32;
  return (void*)(ptr8+offset);
}

void* resultOffset(void* ptr, int32_t bytes) {
  uint8_t* ptr8=(uint8_t*)ptr;

  return (void*)(ptr8+bytes);
}

extern "C" {

void* createContext(uint32_t curve, uint32_t maxBatchCount, uint32_t maxPointCount) {
  MSMBatch* batch=new MSMBatch(curve, maxBatchCount, maxPointCount);

  batch->scalarReadyEvents=(hipEvent_t*)malloc(sizeof(hipEvent_t)*maxBatchCount);
  batch->resultReadyEvents=(hipEvent_t*)malloc(sizeof(hipEvent_t)*maxBatchCount);

  CUDA_CHECK(hipEventCreate(&batch->eighthReadyEvent));
  CUDA_CHECK(hipEventCreate(&batch->halfReadyEvent));
  for(int i=0;i<maxBatchCount;i++) {
    CUDA_CHECK(hipEventCreate(&batch->scalarReadyEvents[i]));
    CUDA_CHECK(hipEventCreate(&batch->resultReadyEvents[i]));
  }
  
  return batch;
}

int32_t destroyContext(void* contextPtr) {
  MSMBatch* batch=(MSMBatch*)contextPtr;

  CUDA_CHECK(hipEventDestroy(batch->eighthReadyEvent));
  CUDA_CHECK(hipEventDestroy(batch->halfReadyEvent));
  for(int i=0;i<batch->maxBatchCount;i++) {
    CUDA_CHECK(hipEventDestroy(batch->scalarReadyEvents[i]));
    CUDA_CHECK(hipEventDestroy(batch->resultReadyEvents[i]));
  }

  // clean up as much memory as possible
  if(batch->pointMemory!=NULL) 
    CUDA_CHECK(hipFree(batch->pointMemory));
  if(batch->scaledPointMemory!=NULL) 
    CUDA_CHECK(hipFree(batch->scaledPointMemory));
  if(batch->scalarMemory!=NULL) 
    CUDA_CHECK(hipFree(batch->scalarMemory));
  if(batch->planningMemory!=NULL)
    CUDA_CHECK(hipFree(batch->planningMemory));
  if(batch->bucketMemory!=NULL)
    CUDA_CHECK(hipFree(batch->bucketMemory));
  if(batch->reduceMemory!=NULL)
    CUDA_CHECK(hipFree(batch->reduceMemory));

  delete batch;
  return 0;
}

int32_t preprocessPoints(void* contextPtr, void* pointData, uint32_t pointCount) {
  MSMBatch*    batch=(MSMBatch*)contextPtr;
  hipStream_t stream;

  // clean up as much memory as possible
  if(batch->scalarMemory!=NULL) 
    CUDA_CHECK(hipFree(batch->scalarMemory));
  batch->scalarMemory=NULL;
  if(batch->planningMemory!=NULL)
    CUDA_CHECK(hipFree(batch->planningMemory));
  batch->planningMemory=NULL;
  if(batch->bucketMemory!=NULL)
    CUDA_CHECK(hipFree(batch->planningMemory));
  batch->planningMemory=NULL;
  if(batch->reduceMemory!=NULL)
    CUDA_CHECK(hipFree(batch->reduceMemory));
  batch->reduceMemory=NULL;

  CUDA_CHECK(hipStreamCreate(&stream));
  CUDA_CHECK(hipMalloc(&batch->pointMemory, 96*pointCount));
  CUDA_CHECK(hipMemcpy(batch->pointMemory, pointData, 96*pointCount, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMalloc(&batch->scaledPointMemory, batch->pointBytesRequired()));
  CUDA_CHECK(batch->runPointPrecompute(stream, batch->scaledPointMemory, batch->pointMemory, pointCount));
  CUDA_CHECK(hipStreamDestroy(stream));

  // we're done with the point memory, clean it up
  CUDA_CHECK(hipFree(batch->pointMemory));
  batch->pointMemory=NULL;

  // reallocate the required scalar memory and planning memory
  CUDA_CHECK(hipMalloc(&batch->scalarMemory, 32*batch->maxBatchCount*batch->maxPointCount));
  CUDA_CHECK(hipMalloc(&batch->planningMemory, batch->planningBytesRequired()));
  CUDA_CHECK(hipMalloc(&batch->bucketMemory, batch->bucketBytesRequired()));
  CUDA_CHECK(hipMalloc(&batch->reduceMemory, batch->maxBatchCount*batch->reduceBytesRequired()));
  return 0;
}

int32_t processBatches(void* contextPtr, void* resultData, void** scalarData, uint32_t batchCount, uint32_t pointCount) {
  MSMBatch*    batch=(MSMBatch*)contextPtr;
  hipStream_t memoryStream, runStream;
  void*        gpuScalars;
  void*        gpuResults;
  void*        cpuScalars;
  void*        cpuResult;
  uint32_t     oneEighth=pointCount>>3;
  uint32_t     threeEighths=3*oneEighth;
  uint32_t     oneHalf=pointCount>>1;
  uint32_t     resultPointCount;

  CUDA_CHECK(hipStreamCreate(&memoryStream));
  CUDA_CHECK(hipStreamCreate(&runStream));

  // QUEUE UP ALL THE MEMORY COPIES

  gpuScalars=batch->scalarMemory;
  cpuScalars=scalarData[0];
  CUDA_CHECK(hipMemcpyAsync(scalarOffset(gpuScalars, 0), scalarOffset(cpuScalars, 0), oneEighth*32, hipMemcpyHostToDevice, memoryStream));
  CUDA_CHECK(hipEventRecord(batch->eighthReadyEvent, memoryStream));
  CUDA_CHECK(hipMemcpyAsync(scalarOffset(gpuScalars, oneEighth), scalarOffset(cpuScalars, oneEighth), threeEighths*32, hipMemcpyHostToDevice, memoryStream));
  CUDA_CHECK(hipEventRecord(batch->halfReadyEvent, memoryStream));
  CUDA_CHECK(hipMemcpyAsync(scalarOffset(gpuScalars, oneHalf), scalarOffset(cpuScalars, oneHalf), oneHalf*32, hipMemcpyHostToDevice, memoryStream));
  CUDA_CHECK(hipEventRecord(batch->scalarReadyEvents[0], memoryStream));

  for(int i=1;i<batchCount;i++) {
    gpuScalars=scalarOffset(batch->scalarMemory, pointCount*i);
    cpuScalars=scalarData[i];
    CUDA_CHECK(hipMemcpyAsync(gpuScalars, cpuScalars, pointCount*32, hipMemcpyHostToDevice, memoryStream));
    CUDA_CHECK(hipEventRecord(batch->scalarReadyEvents[i], memoryStream));
  }

  // QUEUE UP ALL THE RUNS 

  gpuScalars=batch->scalarMemory;
  gpuResults=batch->reduceMemory;
  CUDA_CHECK(hipStreamWaitEvent(runStream, batch->eighthReadyEvent));
  CUDA_CHECK(batch->runPlanning(runStream, batch->planningMemory, gpuScalars, 0, oneEighth));
  CUDA_CHECK(batch->runAccumulate(runStream, batch->bucketMemory, batch->planningMemory, batch->scaledPointMemory));

  CUDA_CHECK(hipStreamWaitEvent(runStream, batch->halfReadyEvent));
  CUDA_CHECK(batch->runPlanning(runStream, batch->planningMemory, gpuScalars, oneEighth, oneHalf));
  CUDA_CHECK(batch->runAccumulate(runStream, batch->bucketMemory, batch->planningMemory, batch->scaledPointMemory, true));

  CUDA_CHECK(hipStreamWaitEvent(runStream, batch->scalarReadyEvents[0]));
  CUDA_CHECK(batch->runPlanning(runStream, batch->planningMemory, gpuScalars, oneHalf, pointCount));
  CUDA_CHECK(batch->runAccumulate(runStream, batch->bucketMemory, batch->planningMemory, batch->scaledPointMemory, true));
  CUDA_CHECK(batch->runReduce(runStream, &resultPointCount, gpuResults, batch->bucketMemory));
  CUDA_CHECK(hipEventRecord(batch->resultReadyEvents[0], runStream));

  for(int i=1;i<batchCount;i++) {
    gpuScalars=scalarOffset(batch->scalarMemory, pointCount*i);
    gpuResults=resultOffset(batch->reduceMemory, batch->reduceBytesRequired()*i); 
    CUDA_CHECK(hipStreamWaitEvent(runStream, batch->scalarReadyEvents[i]));
    CUDA_CHECK(batch->runPlanning(runStream, batch->planningMemory, gpuScalars, pointCount));
    CUDA_CHECK(batch->runAccumulate(runStream, batch->bucketMemory, batch->planningMemory, batch->scaledPointMemory));
    CUDA_CHECK(batch->runReduce(runStream, &resultPointCount, gpuResults, batch->bucketMemory));
    CUDA_CHECK(hipEventRecord(batch->resultReadyEvents[i], runStream));
  }

  // PROCESS THE RESULTS

  cpuResult=(void*)malloc(resultPointCount*192);

  for(int i=0;i<batchCount;i++) {
    gpuResults=resultOffset(batch->reduceMemory, batch->reduceBytesRequired()*i); 
    CUDA_CHECK(hipStreamWaitEvent(memoryStream, batch->resultReadyEvents[i]));
    CUDA_CHECK(hipMemcpyAsync(cpuResult, gpuResults, resultPointCount*192, hipMemcpyDeviceToHost, memoryStream));
    CUDA_CHECK(hipStreamSynchronize(memoryStream));
    batch->runFinalReduce(resultOffset(resultData, 96*i), cpuResult, resultPointCount); 
  }

  free(cpuResult);

  CUDA_CHECK(hipStreamDestroy(runStream));
  CUDA_CHECK(hipStreamDestroy(memoryStream));
  return 0;
}

}